#include "hip/hip_runtime.h"
#include <math.h>
#define M_PI 3.14159265358979323846

__host__ __device__ inline float gaussian(float x, float sigma) {	
	return 1.0f/(sigma*sqrtf(2*M_PI))*expf(-(x*x) / (2 * sigma*sigma));
}

__global__ void gpuBFCalculation(float *input,float *output, float *cGaussian, int height, int width, int kernelWidth, float sigmaR) {	
	//Calculate our pixel's location
	int col=blockIdx.x*blockDim.x + threadIdx.x;	
	int row=blockIdx.y*blockDim.y + threadIdx.y;
	//Boundary check
	if (row >= height || col >= width)
		return;

	float sum = 0;
	float totalWeight = 0;
	float centerIntensity = input[row * width + col];

	for (int dy= -1 * (kernelWidth / 2); dy <= kernelWidth / 2; dy++) {
		for (int dx= -1 * (kernelWidth / 2); dx <= kernelWidth / 2; dx++) {
			if (row + dy < 0 || row + dy >= height || col + dx < 0 || col + dx >= width)
				continue;
			float kernelPosIntensity=input[(row + dy)*width + (col + dx)];			
			float weight= cGaussian[(dy + kernelWidth / 2) * kernelWidth + (dx + kernelWidth / 2)] * gaussian(kernelPosIntensity - centerIntensity, sigmaR);				
			sum+=(weight*kernelPosIntensity);
			totalWeight+=weight;			
		}
	}	
	outimage[row * width + col] = sum / totalWeight;
}

float* BFLaunch(float* input, int width, int height, float sigmaS, float sigmaR){
	int kernelWidth = log2f(min(width, height));
	float* fGaussian =  (float*)malloc((kernelRadius * 2 + 1) * sizeof(float));
	float *d_cGaussian;
	float* output,
	for (int i = 0; i < 2 * kernelRadius + 1; ++i){
		for (int j = 0; j < 2 * kernelRadius + 1; ++i){
			float x = sqrtf((i - kernelRadius) * (i - kernelRadius) + (j - kernelRadius) * (j - kernelRadius));
			fGaussian[i * (2 * kernelRadius + 1)] = gaussian(x, sigmaS);
		}
	}
	hipMalloc(&d_cGaussian, sizeof(float)*(kernelRadius * 2 + 1));
	hipMemcpy(d_cGaussian, fGaussian, sizeof(float)*(kernelRadius*2 + 1), hipMemcpyHostToDevice);
	free(fGaussian);

	float *d_input;
	float *d_output;
	//Cuda memory allocation and error check
	gpuErrchk(hipMalloc(&d_input, sizeof(float)*width*height));//GPU-memory allocation for d_padimage
	gpuErrchk(hipMemcpy(d_input, input, sizeof(float)*width*height, hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc(&d_output, sizeof(float)*width*height));
	dim3 threadsPerBlock(16,16);//normally 16*16 is optimal
	dim3 numBlocks(ceil((float)height / threadsPerBlock.x), ceil((float)width / threadsPerBlock.y)); 
	gpuBFCalculation <<<numBlocks, threadsPerBlock >>> (d_input, d_output, d_cGaussian, height, width, kernelRadius, sigmaR);
	gpuErrchk(hipMemcpy(output, d_output, sizeof(float)*width*height, hipMemcpyDeviceToHost));
	hipFree(d_input);
	hipFree(d_output);
	hipFree(d_cGaussian);

	return outimage;
}