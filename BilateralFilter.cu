#include "hip/hip_runtime.h"
#define M_PI 3.14159265358979323846

__device__ inline float gaussian(float x, float sigma) {	
	return 1.0f/(sigma*sqrtf(2*M_PI))*expf(-(x*x) / (2 * sigma*sigma));
}

__global__ void gpuBFCalculation(float *input,float *output, float *cGaussian, int height, int width, int kernelWidth, float sigmaR) {	
	//Calculate our pixel's location
	int col=blockIdx.x*blockDim.x + threadIdx.x;	
	int row=blockIdx.y*blockDim.y + threadIdx.y;
	//Boundary check
	if (row >= height || col >= width)
		return;

	float sum = 0;
	float totalWeight = 0;
	float centerIntensity = input[row * width + col];

	for (int dy= -1 * (kernelWidth / 2); dy <= kernelWidth / 2; dy++) {
		for (int dx= -1 * (kernelWidth / 2); dx <= kernelWidth / 2; dx++) {
			if (row + dy < 0 || row + dy >= height || col + dx < 0 || col + dx >= width)
				continue;
			float kernelPosIntensity=input[(row + dy)*width + (col + dx)];			
			float weight= cGaussian[(dy + kernelWidth / 2) * kernelWidth + (dx + kernelWidth / 2)] * gaussian(kernelPosIntensity - centerIntensity, sigmaR);				
			sum+=(weight*kernelPosIntensity);
			totalWeight+=weight;			
		}
	}	
	outimage[row * width + col] = sum / totalWeight;
}