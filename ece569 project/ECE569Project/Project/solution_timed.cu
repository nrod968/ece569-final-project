#include "hip/hip_runtime.h"
//
// Noah, Yousuf, Nathaniel
// ECE 569 Project
// 
// 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <wb.h>
#include <png.h>
#include "kernel.cu"
#include "util.cu"

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;

  // PHASE 0
  // Loading the ppm image

  int imageChannels;
  int imageWidth;
  int imageHeight;

  char *inputImageFile;

  wbImage_t inputImage;
  wbImage_t outputImage;

  float *hostInputImageData;
  float *hostOutputImageData;
  
  float *deviceInputImageData;
  float *deviceOutputImageData;
  float *gsData;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);

  // Since the image is monochromatic, it only contains one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float));

  hipMalloc((void **)&gsData,
             imageWidth * imageHeight * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");

  // Define threads per block to be 16 x 16, and number of blocks to accomodate entire image.
  dim3 numBlocks( ceil(imageWidth / 16.0), ceil(imageHeight / 16.0) );
  dim3 tpb(16, 16);

  // Launch colToGray
  colToGray<<<numBlocks, tpb>>>(deviceInputImageData, gsData, imageWidth, imageHeight, imageChannels);

  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  save_image_to_pgm("outb.pgm", outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
