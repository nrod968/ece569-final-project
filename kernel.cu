
#include <hip/hip_runtime.h>
//
// kernel.cu

///////////////////////// colToGray

__global__ void colToGray_v0(float *inImage, float *outImage, int width, int height, int numChannels) {

    // Determine column and row of thread
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    
    // If within bounds of image, perform grayscale operation.
    if (col < width && row < height) {
      int index = (row * width + col);
      int rgbIndex = index * numChannels;
  
      float r = inImage[rgbIndex];
      float g = inImage[rgbIndex + 1];
      float b = inImage[rgbIndex + 2];
  
      outImage[index] = (0.21 * r + 0.71 * g + 0.07 * b);
    }
}

__global__ void colToGray_v1_0(float *inImage, int imageArea) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= imageArea * 3) return;
    inImage[idx] = inImage[idx] * 0.07;
}

__global__ void colToGray_v1_1(float *inImage, float *outImage, int imageArea) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= imageArea) return;

    int rgbIndex = idx * 3;

    float r = inImage[rgbIndex];
    float g = inImage[rgbIndex + 1];
    float b = inImage[rgbIndex + 2];
  
    outImage[idx] = (3 * r + 10 * g + b);
}

__global__ void colToGray_v2(float *inImage, float *outImage, int imageArea) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= imageArea) return;

    int rgbIndex = idx * 3;

    float r = inImage[rgbIndex];
    float g = inImage[rgbIndex + 1];
    float b = inImage[rgbIndex + 2];
  
    outImage[idx] = (0.21 * r + 0.71 * g + 0.07 * b);
}

__global__ void colToGray_v2_byte(float *inImage, uint8_t *outImage, int imageArea) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= imageArea) return;

    int rgbIndex = idx * 3;

    float r = inImage[rgbIndex];
    float g = inImage[rgbIndex + 1];
    float b = inImage[rgbIndex + 2];
  
    outImage[idx] = (uint8_t)((0.21 * r + 0.71 * g + 0.07 * b) * 255);
}

  // num threads = image area * 3
__global__ void colToGray_v3_0(float *inImage, float *intImage, int imageArea) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= imageArea * 3) return;
    int outIdx = (idx % 3) * imageArea + (idx / 3);
    intImage[outIdx] = inImage[idx];
}

__global__ void colToGray_v3_1(float *intImage, float *outImage, int imageArea) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= imageArea) return;

    float r = intImage[idx];
    float g = intImage[idx + imageArea];
    float b = intImage[idx + imageArea + imageArea];
  
    outImage[idx] = (0.21 * r + 0.71 * g + 0.07 * b);
}

__global__ void colToGray_v4(float *inImage, float *outImage, int imageArea) {
    __shared__ float s[1024];

    int i = threadIdx.x;
    int idx = (threadIdx.x + blockIdx.x * blockDim.x) - blockIdx.x;
    if (idx >= imageArea * 3) return;

    s[i] = inImage[idx];

    __syncthreads();

    if (i < 341) {
        int index = i * 3;
        outImage[i + (blockIdx.x * 341)] = (0.21 * s[index] + 0.71 * s[index+1] + 0.07 * s[index+2]);
    }
}

///////////////////////// cannyEdge

__global__ void cannyEdge_v0_0(float *imageIn, float *gradient, float *angle, int width, int height) {

    // 0 1 2
    // 3   4
    // 5 6 7

    float n[8];

    for (int i = 0; i < 8; i++) {
        n[i] = 0;
    }

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (col < width && row < height) {
        int index = (row * width + col);

        if (col > 0) {
            if (row > 0)
                n[0] = imageIn[index - width - 1];

            if (row < height - 1)
                n[5] = imageIn[index + width - 1]; 

            n[3] = imageIn[index - 1];
        }

        if (col < width - 1) {
            if (row > 0)
                n[2] = imageIn[index - width + 1];

            if (row < height - 1)
                n[7] = imageIn[index + width + 1];

            n[4] = imageIn[index + 1];
        }

        if (row > 0) 
            n[1] = imageIn[index - width];

        if (row < height - 1)
            n[6] = imageIn[index + width];

        float gx = -n[0] + n[2] - (2 * n[3]) + (2 * n[4]) - n[5] + n[7];
        float gy = -n[0] + n[5] - (2 * n[1]) + (2 * n[6]) - n[2] + n[7];

        float grad = sqrtf( pow(gx, 2) + pow(gy, 2) );
        float theta = atan2f( gy, gx );
        theta = theta + (theta < 0) * M_PI;

        gradient[index] = grad;
        angle[index] = theta;
    }
}

__global__ void cannyEdge_v1_0(float *imageIn, float *gradient, float *angle, int width, int height) {

    // 0 1 2
    // 3   4
    // 5 6 7

    __shared__ float n[18][18];

    int col = threadIdx.x + blockIdx.x * blockDim.x; // global col
    int row = threadIdx.y + blockIdx.y * blockDim.y; // global row
    
    if (col >= width || row >= height) return;

    for (int i = 0; i < 18; i++) {
        for (int j = 0; j < 18; j++) {
            n[i][j] = 0;
        }
    }

    __syncthreads();

    int bCol = threadIdx.x; // block col
    int bRow = threadIdx.y; // block row

    int index = (row * width + col);

    n[bCol + 1][bRow + 1] = imageIn[index];

    if (bCol == 0 && col > 0)       n[0][bRow + 1] = imageIn[index - 1]; 
    if (bCol == 15 && col < width - 1)  n[17][bRow + 1] = imageIn[index + 1]; 
    if (bRow == 0 && row > 0)       n[bCol + 1][0] = imageIn[index - width];
    if (bRow == 15 && row < height - 1) n[bCol + 1][17] = imageIn[index + width];

    if (bCol == 0 && bRow == 0 && col > 0 && row > 0)            n[0][0] =  imageIn[index - width - 1];
    if (bCol == 15 && bRow == 0 && col < width - 1 && row > 0)       n[17][0] =  imageIn[index - width + 1];
    if (bCol == 0 && bRow == 15 && col > 0 && row < height - 1)      n[0][17] =  imageIn[index + width - 1];
    if (bCol == 15 && bRow == 15 && col < width - 1 && row < height - 1) n[17][17] =  imageIn[index + width + 1];

    __syncthreads();

    float gx = -n[bCol][bRow] + n[bCol+2][bRow] - (2 * n[bCol][bRow+1]) + (2 * n[bCol+2][bRow+1]) - n[bCol][bRow+2] + n[bCol+2][bRow+2];
    float gy = -n[bCol][bRow] + n[bCol][bRow+2] - (2 * n[bCol+1][bRow]) + (2 * n[bCol+1][bRow+2]) - n[bCol+2][bRow] + n[bCol+2][bRow+2];

    float grad = sqrtf( pow(gx, 2) + pow(gy, 2) );
    float theta = atan2f( gy, gx );
    theta = theta + (theta < 0) * M_PI;

    gradient[index] = grad;
    angle[index] = theta;
}

__global__ void cannyEdge_v2_0(float *imageIn, float2 *out, int width, int height) {

    // 0 1 2
    // 3   4
    // 5 6 7

    float n[8];

    for (int i = 0; i < 8; i++) {
        n[i] = 0;
    }

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (col < width && row < height) {
        int index = (row * width + col);

        if (col > 0) {
            if (row > 0)
                n[0] = imageIn[index - width - 1];

            if (row < height - 1)
                n[5] = imageIn[index + width - 1]; 

            n[3] = imageIn[index - 1];
        }

        if (col < width - 1) {
            if (row > 0)
                n[2] = imageIn[index - width + 1];

            if (row < height - 1)
                n[7] = imageIn[index + width + 1];

            n[4] = imageIn[index + 1];
        }

        if (row > 0) 
            n[1] = imageIn[index - width];

        if (row < height - 1)
            n[6] = imageIn[index + width];

        float gx = -n[0] + n[2] - (2 * n[3]) + (2 * n[4]) - n[5] + n[7];
        float gy = -n[0] + n[5] - (2 * n[1]) + (2 * n[6]) - n[2] + n[7];

        float grad = pow(gx, 2) + pow(gy, 2);
        float theta = atan2f( gy, gx );
        theta = theta + (theta < 0) * M_PI;

        float2 val = make_float2(grad, theta);

        out[index] = val;
    }
}

__global__ void cannyEdge_v2_1(float2 *in, float *edgemap, int width, int height, float lowThresh) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (col < width && row < height) {
        int index = (row * width + col);

        float grad = in[index].x;
        float theta = in[index].y;

        float max = 1;

        if ( (theta < M_PI / 8) || (theta > (7 * M_PI / 8)) ) {
            if (col > 0 && in[index - 1].x > grad)
                max = 0;
            if (col < width - 1 && in[index + 1].x > grad)
                max = 0;
        }
        else if ( theta < (3 * M_PI / 8) ) {
            if (row > 0 && col > 0 && in[index - width - 1].x > grad)
                max = 0;
            if (row < height - 1 && col < width - 1 && in[index + width + 1].x > grad)
                max = 0;
        }
        else if ( theta < (5 * M_PI / 8) ) {
            if (row > 0 && in[index - width].x > grad)
                max = 0;
            if (row < height - 1 && in[index + width].x > grad)
                max = 0;
        }
        else {
            if (row > 0 && col < width - 1 && in[index - width + 1].x > grad)
                max = 0;
            if (row < height - 1 && col > 0 && in[index + width - 1].x > grad)
                max = 0;
        }

        if (grad < pow(lowThresh, 2)) {
            max = 0;
        }

        edgemap[index] = max;
    }
}

__global__ void cannyEdge_v0_1(float *gradient, float *angle, float *edgemap, int width, int height, float lowThresh) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (col < width && row < height) {
        int index = (row * width + col);

        float grad = gradient[index];
        float theta = angle[index];

        float max = 1;

        if ( (theta < M_PI / 8) || (theta > (7 * M_PI / 8)) ) {
            if (col > 0 && gradient[index - 1] > grad)
                max = 0;
            if (col < width - 1 && gradient[index + 1] > grad)
                max = 0;
        }
        else if ( theta < (3 * M_PI / 8) ) {
            if (row > 0 && col > 0 && gradient[index - width - 1] > grad)
                max = 0;
            if (row < height - 1 && col < width - 1 && gradient[index + width + 1] > grad)
                max = 0;
        }
        else if ( theta < (5 * M_PI / 8) ) {
            if (row > 0 && gradient[index - width] > grad)
                max = 0;
            if (row < height - 1 && gradient[index + width] > grad)
                max = 0;
        }
        else {
            if (row > 0 && col < width - 1 && gradient[index - width + 1] > grad)
                max = 0;
            if (row < height - 1 && col > 0 && gradient[index + width - 1] > grad)
                max = 0;
        }

        if (grad < lowThresh) {
            max = 0;
        }

        edgemap[index] = max;
    }
}

__global__ void cannyEdge_v3(float *imageIn, float *edgemap, int width, int height, int lowThresh) {

    // // 0 1 2
    // // 3   4
    // // 5 6 7

    // Part 1: calculate gradient and angle

    __shared__ int ns[64][8];

    int col = (threadIdx.x + blockIdx.x * blockDim.x) - (4 * blockIdx.x) - 2;
    int row = (threadIdx.y + blockIdx.y * blockDim.y) - (4 * blockIdx.y) - 2;
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int index = row * width + col;

    if (col >= 0 && row >= 0 && col < width && row < height)
        ns[tidx][tidy] = (int)(imageIn[index] * 100);
    else
        ns[tidx][tidy] = 0;

    __syncthreads();

    if (tidx >= 1 && tidy >= 1 && tidx <= blockDim.x-2 && tidy <= blockDim.y-2) {

        int gx = -ns[tidx-1][tidy-1] + ns[tidx+1][tidy-1] - (2 * ns[tidx-1][tidy]) + (2 * ns[tidx+1][tidy]) - ns[tidx-1][tidy+1] + ns[tidx+1][tidy+1];
        int gy = -ns[tidx-1][tidy-1] + ns[tidx-1][tidy+1] - (2 * ns[tidx][tidy-1]) + (2 * ns[tidx][tidy+1]) - ns[tidx+1][tidy-1] + ns[tidx+1][tidy+1];

        __syncthreads();

        int grad = (int)(sqrtf( powf(gx, 2) + powf(gy, 2) ));
        int theta = (int)(atan2f( gy, gx ) * 180 / M_PI);
        theta = theta + (theta < 0) * 180;

        ns[tidx][tidy] = grad;

        __syncthreads();

        // Part 2: find edges

        if (tidx >= 2 && tidy >= 2 && tidx <= blockDim.x-3 && tidy <= blockDim.y-3) {    
            float max = 1;

            if (grad < lowThresh) {
                max = 0;
            }
    
            if ( (theta < 22) || (theta > 157) ) {
                if (ns[tidx-1][tidy] > grad)
                    max = 0;
                if (ns[tidx+1][tidy] > grad)
                    max = 0;
            }
            else if ( theta < 67 ) {
                if (ns[tidx-1][tidy-1] > grad)
                    max = 0;
                if (ns[tidx+1][tidy+1] > grad)
                    max = 0;
            }
            else if ( theta < 112 ) {
                if (ns[tidx][tidy-1] > grad)
                    max = 0;
                if (ns[tidx][tidy+1] > grad)
                    max = 0;
            }
            else {
                if (ns[tidx+1][tidy-1] > grad)
                    max = 0;
                if (ns[tidx-1][tidy+1] > grad)
                    max = 0;
            }
    
            if (col < width && row < height)
                edgemap[index] = max;
            
        }
    }
}

__device__ float fastatan2f(float a, float b) {
    if (fabs(b - 0.0001f) > 0)
        b = 0.001f;
    double x = a / b;
    x = x * x;
    return (float)((0.077650 * x - 0.287434) * x + 0.9951816) * x;
}

__device__ float fastsinf(float a) {
    return a;
}

__device__ float fastcosf(float a) {
    return -0.4 * a * a + 1;
}

__global__ void cannyEdge_v4(float *imageIn, float *edgemap, int width, int height, int lowThresh) {
    // Part 1: calculate gradient and angle

    __shared__ uint16_t ns[16][32];
    __shared__ uint16_t grads[16][32];

    int col = (threadIdx.x + blockIdx.x * blockDim.x) - (4 * blockIdx.x) - 2;
    int row = (threadIdx.y + blockIdx.y * blockDim.y) - (4 * blockIdx.y) - 2;
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int index = row * width + col;

    if (col >= 0 && row >= 0 && col < width && row < height)
        ns[tidx][tidy] = (uint16_t)(imageIn[index] * 100);
    else
        ns[tidx][tidy] = 0;

    __syncthreads();

    if (tidx >= 1 && tidy >= 1 && tidx <= blockDim.x-2 && tidy <= blockDim.y-2 && col < width && row < height) {

        int gx = (int)ns[tidx+1][tidy] - (int)ns[tidx-1][tidy];
        int gy = (int)ns[tidx][tidy+1] - (int)ns[tidx][tidy-1];

        uint16_t grad = ( gx * gx + gy * gy );
        float theta = fastatan2f( gy, gx );

        grads[tidx][tidy] = grad;

        __syncthreads();

        // Part 2: find edges

        if (tidx >= 2 && tidy >= 2 && tidx <= blockDim.x-3 && tidy <= blockDim.y-3 && grad >= lowThresh) {    
            int x = (int)(fastcosf(theta) - 0.6) + (int)(fastcosf(theta) + 0.6);
            int y = (int)(fastsinf(theta) - 0.6) + (int)(fastsinf(theta) + 0.6);

            int max = fmaxf(grads[tidx + x][tidy + y], grad);
            max = fmaxf( grads[tidx - x][tidy - y], max );
            edgemap[index] = (max == grad);
        }
    }
}

__global__ void cannyEdge_v3_byte(uint8_t *imageIn, uint8_t *edgemap, int width, int height, int lowThresh) {

    // // 0 1 2
    // // 3   4
    // // 5 6 7

    // Part 1: calculate gradient and angle

    __shared__ int ns[32][8];

    int col = (threadIdx.x + blockIdx.x * blockDim.x) - (4 * blockIdx.x) - 2;
    int row = (threadIdx.y + blockIdx.y * blockDim.y) - (4 * blockIdx.y) - 2;
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int index = row * width + col;

    if (col >= 0 && row >= 0 && col < width && row < height)
        ns[tidx][tidy] = (int)(imageIn[index]);
    else
        ns[tidx][tidy] = 0;

    __syncthreads();

    if (tidx >= 1 && tidy >= 1 && tidx <= blockDim.x-2 && tidy <= blockDim.y-2) {

        int gx = -ns[tidx-1][tidy-1] + ns[tidx+1][tidy-1] - (2 * ns[tidx-1][tidy]) + (2 * ns[tidx+1][tidy]) - ns[tidx-1][tidy+1] + ns[tidx+1][tidy+1];
        int gy = -ns[tidx-1][tidy-1] + ns[tidx-1][tidy+1] - (2 * ns[tidx][tidy-1]) + (2 * ns[tidx][tidy+1]) - ns[tidx+1][tidy-1] + ns[tidx+1][tidy+1];

        int grad = (int)(sqrtf( powf(gx, 2) + powf(gy, 2) ));
        int theta = (int)(atan2f( gy, gx ) * 180 / M_PI);
        theta = theta + (theta < 0) * 180;

        __syncthreads();

        ns[tidx][tidy] = grad;

        __syncthreads();

        // Part 2: find edges
    
        if (tidx >= 2 && tidy >= 2 && tidx <= blockDim.x-3 && tidy <= blockDim.y-3) {    
            uint8_t max = 1;
    
            if ( (theta < 22) || (theta > 157) ) {
                if (ns[tidx-1][tidy] > grad)
                    max = 0;
                if (ns[tidx+1][tidy] > grad)
                    max = 0;
            }
            else if ( theta < 67 ) {
                if (ns[tidx-1][tidy-1] > grad)
                    max = 0;
                if (ns[tidx+1][tidy+1] > grad)
                    max = 0;
            }
            else if ( theta < 112 ) {
                if (ns[tidx][tidy-1] > grad)
                    max = 0;
                if (ns[tidx][tidy+1] > grad)
                    max = 0;
            }
            else {
                if (ns[tidx+1][tidy-1] > grad)
                    max = 0;
                if (ns[tidx-1][tidy+1] > grad)
                    max = 0;
            }
    
            if (grad < lowThresh) {
                max = 0;
            }
    
            if (col < width && row < height)
                edgemap[index] = max;
            
        }
    }
}

///////////////////////// applyMask
__global__ void applyMask_v0(float* inEdgemap, float* outMasked, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= width || idy >= height) return;

    // Define the points of the triangle (bottom left corner, bottom right corner, center of image)
    float x1 = 0;
    float y1 = height - 1;

    float x2 = width - 1;
    float y2 = height - 1;

    float x3 = (float)width / 2;
    float y3 = (float)height / 2;

    // Compute the barycentric coordinates of the current pixel
    float alpha = ((y2 - y3)*(idx - x3) + (x3 - x2)*(idy - y3)) / ((y2 - y3)*(x1 - x3) + (x3 - x2)*(y1 - y3));
    float beta = ((y3 - y1)*(idx - x3) + (x1 - x3)*(idy - y3)) / ((y2 - y3)*(x1 - x3) + (x3 - x2)*(y1 - y3));
    float gamma = 1.0 - alpha - beta;

    // Check if the current pixel is inside the triangle
    if (alpha >= 0.0 && beta >= 0.0 && gamma >= 0.0 && inEdgemap[idy * width + idx] > 0.5f) {
        outMasked[idy * width + idx] = 1.0;
    } else {
        outMasked[idy * width + idx] = 0.0;
    }
}

__global__ void applyMask_v1(float* inEdgemap, float* outMasked, int width, int height,
                               float wX, float hY, float denom) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= width || idy >= height) return;

    // Compute variables & global index
    float xA = idx - (wX + 1);
    float yA = idy - (hY + 1);
    int index = idy * width + idx;

    // Compute the barycentric coordinates of the current pixel
    float alpha = (hY * xA - wX * yA) / denom;
    float beta =  (-hY * xA - (wX+1) * yA) / denom;
    float gamma = 1.0 - alpha - beta;

    // Check if the current pixel is inside the triangle
    outMasked[index] = (alpha >= 0.0 && beta >= 0.0 && gamma >= 0.0 && inEdgemap[index] > 0.5f);
}

__global__ void applyMask_v1_byte(uint8_t* inEdgemap, float* outMasked, int width, int height,
    float wX, float hY, float denom) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int idy = blockIdx.y * blockDim.y + threadIdx.y;
if (idx >= width || idy >= height) return;

// Compute variables & global index
float xA = idx - (wX + 1);
float yA = idy - (hY + 1);
int index = idy * width + idx;

// Compute the barycentric coordinates of the current pixel
float alpha = (hY * xA - wX * yA) / denom;
float beta =  (-hY * xA - (wX+1) * yA) / denom;
float gamma = 1.0 - alpha - beta;

// Check if the current pixel is inside the triangle
outMasked[index] = (alpha >= 0.0 && beta >= 0.0 && gamma >= 0.0 && inEdgemap[index] == 1);
}

/////////////////////////////////// Hough

// theta increment, total of 17 bins from 0 to pi
#define N_ROWS 16
#define DELTA_THETA M_PI / N_ROWS
#define DELTA_RHO 2.0

__host__ int findNCols(int width, int height) {
    int N = height;
    if (width > height)
        N = width;

    return (int) ( N * 2.414 / DELTA_RHO );
}

__device__ __host__ int getRhoIndex(float rho, int ncols) {
    return (int)floorf( (rho + ncols / DELTA_RHO) + 0.5 );
}

__device__ float getRho(float theta, int idx, int idy) {
    return idx * cosf(theta) + idy * sinf(theta);
}

// (x1, y1) is LEFTmost point of the line
// (x2, y2) is the RIGHTmost point of the line
// x1
// y1
// x2
// y2
// slope

// Writes into the hough array
__global__ void hough_v0_0(float *inMasked, int width, int height, int *hArray, int ncols, int *xMins, int *yMins, int *xMaxs, int *yMaxs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idy * width + idx;
    int hArraySize = N_ROWS * ncols;

    if (idx >= width || idy >= height || inMasked[index]  < 0.5f) return;

    for (int i = 0; i < N_ROWS; i++) {
        float theta = i * DELTA_THETA;
        float rho = getRho(theta, idx, idy);
        int rhoIndex = getRhoIndex(rho, ncols);
        index = i * ncols + rhoIndex;

        if (index >= 0 && index < hArraySize) {
            atomicAdd( &(hArray[ index ]), 1  );
            atomicMin( &(xMins[ index ]), idx );
            atomicMin( &(yMins[ index ]), idy );
            atomicMax( &(xMaxs[ index ]), idx );
            atomicMax( &(yMaxs[ index ]), idy );
        }
    }
}

// This one reads from an array that has the precalculated cos(theta) and sin(theta) values. 
// Ignores theta = 0 & pi. Only positive and negative slopes. 
// TODO. nothing has been edited yet.
__global__ void hough_v2_0(float *inMasked, int width, int height, int *hArray, int ncols, int *xMins, int *yMins, int *xMaxs, int *yMaxs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idy * width + idx;
    int hArraySize = N_ROWS * ncols;

    if (idx >= width || idy >= height || inMasked[index]  < 0.5f) return;

    for (int i = 0; i < N_ROWS; i++) {
        float theta = i * DELTA_THETA;
        float rho = getRho(theta, idx, idy);
        int rhoIndex = getRhoIndex(rho, ncols);
        index = i * ncols + rhoIndex;

        if (index >= 0 && index < hArraySize) {
            atomicAdd( &(hArray[ index ]), 1  );
            atomicMin( &(xMins[ index ]), idx );
            atomicMin( &(yMins[ index ]), idy );
            atomicMax( &(xMaxs[ index ]), idx );
            atomicMax( &(yMaxs[ index ]), idy );
        }
    }
}

// Instead of one thread per pixel, this kernel uses one thread per bin in hough array.
// IDEA if this works: tiling with shared memory
__global__ void hough_v1_0(float *inMasked, int width, int height, int *hArray, int ncols, int *xMins, int *yMins, int *xMaxs, int *yMaxs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx >= ncols || idy >= N_ROWS) return;

    float theta = idy * DELTA_THETA;
    float cosTheta = cosf(theta);
    float sinTheta = sinf(theta);

    int count = 0;
    int minX = width;
    int minY = height;
    int maxX = 0;
    int maxY = 0;

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            if (inMasked[i * width + j] == 1) {
                float rho = j * cosTheta + i * sinTheta;
                int rhoIndex = getRhoIndex(rho, ncols);

                if (rhoIndex == idx) {
                    count++;
                    minX = fminf(minX, j);
                    minY = fminf(minY, i);
                    maxX = fminf(maxX, j);
                    maxY = fminf(maxY, i);
                }
            }
        }
    }

    int index = idy * ncols + idx;
    hArray[index] = count;
    xMins[index] = minX;
    yMins[index] = minY;
    xMaxs[index] = maxX;
    yMaxs[index] = maxY;
}

// // Finds peaks and stores back into hArray.
__global__ void hough_v0_1(int *hArray, int ncols, int thresh) {

    __shared__ int hs[16][16];

    int col = (threadIdx.x + blockIdx.x * blockDim.x) - (2 * blockIdx.x) - 1;
    int row = (threadIdx.y + blockIdx.y * blockDim.y) - (2 * blockIdx.y) - 1;
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int index = row * ncols + col;

    if (col >= 0 && row >= 0 && col < ncols && row < N_ROWS)
        hs[tidx][tidy] = hArray[index];
    else
        hs[tidx][tidy] = 0;

    __syncthreads();

    if (tidx >= 1 && tidy >= 1 && tidx <= blockDim.x-2 && tidy <= blockDim.y-2 && col < ncols && row < N_ROWS) {
        int peak = 0;

        int h = hs[tidx][tidy];
        if ( h >= thresh && h >= hs[tidx-1][tidy] && h >= hs[tidx+1][tidy] && h >= hs[tidx][tidy-1] && h >= hs[tidx][tidy+1] ) {
            peak = 1;
        }
        hArray[index] = peak;
    }
}

#define MAX_LINES 32

// Running sum and then writes the first MAX_LINES lines to the lines array.
//
// 
//
//
// __global__ void hough_v1_2(int *hArray, int ncols, int *sums, int *xMins, int *yMins, int *xMaxs, int *yMaxs, int ncols, float *lines, int numBlocks) {

//     __shared__ int xy[2*32];
//     int inputSize = ncols * N_ROWS;

//     int i = threadIdx.x + blockIdx.x * blockDim.x;
//     int tid = threadIdx.x;
    
//     if (i < inputSize)
//         xy[tid] = hArray[i];

//     int stride;
//     for (stride = 1; stride <= blockDim.x; stride *= 2) {
//         __syncthreads();
//         int index = (tid + 1) * stride * 2 - 1;
//         if (index < 2 * blockDim.x)
//             xy[index] += xy[index - stride];
//     }

//     __syncthreads();

//     for (stride = blockDim.x / 2; stride > 0; stride /= 2) {
//         __syncthreads();
//         int index = (tid + 1) * stride * 2 - 1;
//         if (index + stride < blockDim.x)
//             xy[index+stride] += xy[index];
//     }

//     __syncthreads();

//     if (i < inputSize && (tid+1 == BlockDim.x || tid+1 == inputSize))
//         sums[blockIdx.x] = xy[tid];

//     __syncthreads();

//     if (i == 0)
//         houghSumScan<<<ceil(numBlocks / 256.0), 256>>>(sums);

//     __syncthreads();

//     if (i < inputSize && blockIdx.x > 0)
//         xy[tid] += sums[blockIdx.x-1];
       
//     if (i < inputSize)
//         hArray[i] = xy[tid];

//     /// TODO FINISH CODE.

// }

// __global__ void houghSumScan(int *sums) {

//     int i = threadIdx.x + blockIdx.x * blockDim.x;
    
//     int stride;
//     for (stride = 1; stride <= i; stride *= 2) {
//         __syncthreads();
//         int temp = sums[i-stride];
//         __syncthreads();
//         sums[i] += temp;
//     }

// }

// Write into new lines array, up to max number of lines.
// Note that this is run on the HOST, NOT the device.
__host__ void hough_v0_2(int *hPeaks, int *xMins, int *yMins, int *xMaxs, int *yMaxs, int ncols, float *lines, int *numLines) {
    int currLine = 0;

    for (int i = 0; i < N_ROWS; i++) {
        for (int j = 0; j < ncols; j++) {
            int index = i * ncols + j;

            // If peak
            if (hPeaks[index] == 1) {
                int x1 = xMins[index];
                int x2 = xMaxs[index];
                int y1 = yMins[index];
                int y2 = yMaxs[index];

                // Negative slope
                float theta = i * DELTA_THETA;
                float rho = x1 * cosf(theta) + y1 * sinf(theta);
                int rhoIndex = getRhoIndex(rho, ncols);

                // If equal, then slope is negative.
                if (rhoIndex == j) {
                    lines[currLine                ] = x1;
                    lines[currLine +     MAX_LINES] = y1;
                    lines[currLine + 2 * MAX_LINES] = x2;
                    lines[currLine + 3 * MAX_LINES] = y2;
                    lines[currLine + 4 * MAX_LINES] = (y2 - y1) / ((float)(x2 - x1) + 0.001);
                }
                // Otherwise, slope is positive and y values are flipped.
                else {
                    lines[currLine                ] = x1;
                    lines[currLine +     MAX_LINES] = y2;
                    lines[currLine + 2 * MAX_LINES] = x2;
                    lines[currLine + 3 * MAX_LINES] = y1;
                    lines[currLine + 4 * MAX_LINES] = (y1 - y2) / ((float)(x2 - x1) + 0.001);
                }

                currLine++;
                if (currLine >= MAX_LINES) {
                    *numLines = currLine;
                    return;
                }
                    
            }
        }
    }
    *numLines = currLine;
}